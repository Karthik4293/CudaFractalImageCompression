#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "cudaCompressor.h"
#include "image.h"

struct GlobalConstants {
    int imageWidth;
    int imageHeight;
    float* imageData;
};

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

CudaCompressor::CudaCompressor(const std::string& imageFilename, int rangeSize, int domainSize) {
}

CudaCompressor::~CudaCompressor() {
}

void CudaCompressor::compress() {

}

void CudaCompressor::saveToFile(const std::string& filename) {

}

CompressedImage* CudaCompressor::getCompressedContents() {
    return NULL;
}